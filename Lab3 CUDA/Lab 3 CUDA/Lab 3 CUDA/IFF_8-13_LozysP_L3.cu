#include "hip/hip_runtime.h"

#include <fstream>
#include <iostream>

class Student
{
public:
	Student() {}
	Student(std::string _name, int year, float grade, char gender)
		: year(year), grade(grade), gender(gender)
	{
		strcpy(name, _name.c_str());
	}
	
	char name[20];
	int year;
	float grade;
	char gender;
};

void read_data (const char* filePath, Student students[]) {

	std::ifstream fin(filePath);

	size_t index = 0;
	while (!fin.eof())
	{
		std::string name;
		int year;
		float grade;
		char gender;
		fin >> name >> year >> grade >> gender >> std::ws;

		students[index++] = Student(name, year, grade, gender);
	}
}

__global__ void process_data(Student *device_students,char *device_results, int* result_space);

int main() {

	Student students[1000];
	read_data("data1.csv", students);	

	int result_space = 50;

	// Result string
	char *host_results = new char[sizeof(char) * 100 * 1000];

	// Allocate GPU memory
	Student *device_students;
	char *device_results;
	int *device_result_space;

	hipMalloc((void**) &device_results		, sizeof(char) * result_space * 1000);
	hipMalloc((void**) &device_students	, sizeof(Student) * 1000);
	hipMalloc((void**) &device_result_space, sizeof(int));

	// Copy from CPU to GPU data that is needed
	hipMemcpy(device_students, 	&students[0],   sizeof(Student) * 1000			  , hipMemcpyHostToDevice);
	hipMemcpy(device_result_space, &result_space,  sizeof(int)						  , hipMemcpyHostToDevice);

	// Run
	process_data<<<1, 70>>>(device_students, device_results, device_result_space);
	hipDeviceSynchronize();


	auto err = hipMemcpy(host_results, device_results, sizeof(char) * result_space * 1000, hipMemcpyDeviceToHost); // copy students to GPU
	std::cout << "Copy to host "<< err << std::endl;
	std::cout << "Result: \n"<< host_results << std::endl;


	// Fee CPU and GPU memory
	free(host_results);
	hipFree(device_results);
	hipFree(device_students);
	hipFree(device_result_space);
}

__global__ void process_data(Student *device_students,char *device_results, int* result_space) {

	auto name = device_students[threadIdx.x].name;
	int offset = threadIdx.x * (*result_space);

	bool name_ended = false;
	for (size_t i = 0; i < *result_space; i++)
	{
		if (name[i] == '\0')
			name_ended = true;
		device_results[i + offset] = name_ended ? ' ' : name[i];
	}
}